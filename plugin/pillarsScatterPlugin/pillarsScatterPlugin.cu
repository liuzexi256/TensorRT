#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_fp16.h>

#include "pillarsScatterPlugin.h"

using namespace nvinfer1;
using nvinfer1::plugin::PillarsScatterPlugin;

///// PillarScatter Enqueue start
template <typename Data>
__global__
void pillar_scatter_kernel(
            int batchSize,
            Data const *idata1,
            Data const *idata2,
            Data *odata,
            const int nDims,
            const int nChans,
            const int _size_h,
            const int _size_w)
{
  const int output_size = _size_h * _size_w * nChans;
  Data const *feat_array = &idata1[nDims * nChans * blockIdx.z];
  Data const *index_array = &idata2[nDims * 4 * blockIdx.z];
  Data *output_array = &odata[output_size * blockIdx.z];
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  for (int index = tid; index < nDims; index += stride)
  {
    for (int c = 0; c < nChans; c++)
    {
      int feature_index =  index * nChans + c;
      int x = (int)index_array[index * 4 + 2];
      int y = (int)index_array[index * 4 + 3];
      int odata_index = c * _size_h * _size_w + x * _size_w + y;
      output_array[odata_index] = feat_array[feature_index];
    }
  }
}

int PillarsScatterPlugin::enqueue(
     int batchSize, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
 {
  //std::cout << "start enqueue pillars scatter" << std::endl;

  int in_feature_dims = 10000;
  int in_channel = 64;
  const dim3 phnetDim3(512, 1, batchSize);

  // if (inputDesc[0].type == nvinfer1::DataType::kFLOAT)
  if (1)
  {
    hipMemsetAsync(outputs[0], 0, sizeof(float) * in_feature_dims * in_channel, stream);
    pillar_scatter_kernel<<<2, phnetDim3, 0, stream>>>
    (
      batchSize,
      static_cast<float const *>(inputs[0]),
      static_cast<float const *>(inputs[1]),
      static_cast<float *>(outputs[0]),
      in_feature_dims,
      in_channel,
      h,
      w
    );
  }
  else
  {
    hipMemsetAsync(outputs[0], 0xFF, sizeof(__half) * in_feature_dims * in_channel, stream);
    pillar_scatter_kernel<<<2, phnetDim3, 0, stream>>>
    (
      batchSize,
      static_cast<__half const *>( inputs[0]),
      static_cast<__half const *>( inputs[1]),
      static_cast<__half *>(outputs[0]),
      in_feature_dims,
      in_channel,
      h,
      w
    );
  }
  return hipGetLastError() != hipSuccess;
}
///// PillarScatter Enqueue end