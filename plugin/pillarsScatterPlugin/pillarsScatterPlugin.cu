#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include "pillarsScatterPlugin.h"

using namespace nvinfer1;
using nvinfer1::plugin::PillarsScatterPlugin;

template <typename Data>
__global__
void pillar_scatter_kernel(
            int batchSize,
            Data const *idata1,
            Data const *idata2,
            Data *odata,
            const int nDims,
            const int nChans,
            const int _size_h,
            const int _size_w)
{
  const int output_size = _size_h * _size_w * nChans;
  Data const *feat_array = &idata1[nDims * nChans * blockIdx.z];
  Data const *index_array = &idata2[nDims * 4 * blockIdx.z];
  Data *output_array = &odata[output_size * blockIdx.z];
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  for (int index = tid; index < nDims; index += stride)
  {
    for (int c = 0; c < nChans; c++)
    {
      int feature_index =  index * nChans + c;
      int x = (int)index_array[index * 4 + 2];
      int y = (int)index_array[index * 4 + 3];
      int odata_index = c * _size_h * _size_w + x * _size_w + y;
      output_array[odata_index] = feat_array[feature_index];
    }
  }
}

int PillarsScatterPlugin::enqueue(const nvinfer1::PluginTensorDesc* inputDesc, const nvinfer1::PluginTensorDesc* outputDesc,
                         const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
  // std::cout << "start enqueue pillars scatter" << std::endl;
  auto const& input0_dims = inputDesc[0].dims;

  const int in_feature_dims = input0_dims.d[1];
  const int in_channel = input0_dims.d[2];
  const int batchSize = inputDesc[0].dims.d[0];
  const dim3 phnetDim3(512, 1, batchSize);

  if (inputDesc[0].type == nvinfer1::DataType::kFLOAT)
  {
    hipMemsetAsync(outputs[0], 0, sizeof(float) * in_feature_dims * in_channel, stream);
    pillar_scatter_kernel<<<2, phnetDim3, 0, stream>>>
    (
      batchSize,
      static_cast<float const *>(inputs[0]),
      static_cast<float const *>(inputs[1]),
      static_cast<float *>(outputs[0]),
      in_feature_dims,
      in_channel,
      _size_h,
      _size_w
    );
  }
  else
  {
    hipMemsetAsync(outputs[0], 0xFF, sizeof(__half) * in_feature_dims * in_channel, stream);
    pillar_scatter_kernel<<<2, phnetDim3, 0, stream>>>
    (
      batchSize,
      static_cast<__half const *>( inputs[0]),
      static_cast<__half const *>( inputs[1]),
      static_cast<__half *>(outputs[0]),
      in_feature_dims,
      in_channel,
      _size_h,
      _size_w
    );
  }
  return hipGetLastError() != hipSuccess;
}