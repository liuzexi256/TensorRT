#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_fp16.h>

#include "gather4DPlugin.h"

using namespace nvinfer1;
using nvinfer1::plugin::Gather4DPlugin;

///// Gather4D Enqueue start
template <typename Data>
__global__
void pillar_map_kernel(
            int batchSize,
            Data const* idata1,
            Data const* idata2,
            Data*       odata,
            int nDims,
            int nChans,
            int _size_h,
            int _size_w
            )
{
  Data const *feat_array = &idata1[nChans * _size_h * _size_w * blockIdx.z];
  Data const *index_array = &idata2[nDims * blockIdx.z];
  Data *output_array = &odata[nDims * nChans * blockIdx.z];
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;

  for (int index = tid; index < nDims; index += stride)
  {
    for (int c = 0; c < nChans; c++)
    {
      int output_index = index * nChans + c;
      int x = (int)index_array[index * 4 + 2];
      int y = (int)index_array[index * 4 + 3];
      int feat_index = c * _size_h * _size_w + x * _size_w + y;
      output_array[output_index] = feat_array[feat_index];
    }
  }
}

int Gather4DPlugin::enqueue(
     int batchSize, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
 {
  //std::cout << "start enqueue gather 4d" << std::endl;

  int in_feature_dims = 10000;
  int in_channel = 96;
  int _size_h = 96;
  int _size_w = 96;
  const dim3 phnetDim3(512, 1, batchSize);

  // if (inputDesc[0].type == nvinfer1::DataType::kFLOAT)
  if (1)
  {
    hipMemsetAsync(outputs[0], 0, sizeof(float) * in_feature_dims * in_channel, stream);
    pillar_map_kernel<<<2, phnetDim3, 0, stream>>>
    (
        batchSize,
        static_cast<float const *>(inputs[0]),
        static_cast<float const *>(inputs[1]),
        static_cast<float *>(outputs[0]),
        in_feature_dims,
        in_channel,
        _size_h,
        _size_w
    );
  }
  else
  {
    hipMemsetAsync(outputs[0], 0, sizeof(__half) * in_feature_dims * in_channel, stream);
    pillar_map_kernel<<<2, phnetDim3, 0, stream>>>
    (
      batchSize,
      static_cast<__half const *>(inputs[0]),
      static_cast<__half const *>(inputs[1]),
      static_cast<__half *>(outputs[0]),
      in_feature_dims,
      in_channel,
      _size_h,
      _size_w
    );
  }
  return hipGetLastError() != hipSuccess;
}
///// Gather4D Enqueue end